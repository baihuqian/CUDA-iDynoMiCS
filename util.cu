#include "hip/hip_runtime.h"
/*
 * util.cu
 *
 *  Created on: Jul 16, 2014
 *      Author: bqian
 */

#include "util.cuh"

void allocateArray(void **devPtr, size_t size)
{
	CUDA_CHECK_RETURN(hipMalloc(devPtr, size));
}


void freeArray(void *devPtr)
{
	CUDA_CHECK_RETURN(hipFree(devPtr));
}

void copyArrayToDevice(void *device, const void *host, int offset, int size)
{
	CUDA_CHECK_RETURN(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

void copyArrayFromDevice(void *host, const void *device, int size)
{
	CUDA_CHECK_RETURN(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
}

void setArray(void *devPtr, int value, int count)
{
	CUDA_CHECK_RETURN(hipMemset(devPtr, value, count));
}

void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = iDivUp(n, numThreads);
}

uint inline iDivUp(uint a, uint b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}
